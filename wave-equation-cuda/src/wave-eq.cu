#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <malloc.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

#define N 1000 // Tamanho da Malha
#define tempoTotal 15000 // Loops temporais
#define	alpha 0.4
#define	gamma 0.8
#define xInicial 0
#define xFinal 50.0
#define yInicial 0
#define yFinal 50.0

#define CHECK(call)                                                \
    {                                                              \
        hipError_t error = call;                                  \
        if (error != hipSuccess)                                  \
        {                                                          \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
        }                                                          \
    }

__global__ void waveEquationKernel(double *wave, double *waveFuture, double *wavePast)
{
    int i = blockIdx.x;
    int j = threadIdx.x;

    if ((i > 0 && i < N - 1) && (j > 0 && j < N - 1))
        waveFuture[i * blockDim.x + j] = 2 * wave[i * blockDim.x + j] * (1 - alpha * alpha - gamma * gamma) - wavePast[i * blockDim.x + j] + alpha * alpha * wave[(i + 1) * blockDim.x + j] + alpha * alpha * wave[(i - 1) * blockDim.x + j] + gamma * gamma * wave[i * blockDim.x + (j + 1)] + gamma * gamma * wave[i * blockDim.x + (j - 1)];
}

void writeFiles(double *wave, double dx, double dy)
{
    int i, j;
    FILE *fileStaticPlot;

    fileStaticPlot = fopen("WaveStatic2.dat", "w");

    fprintf(fileStaticPlot, "x\ty\tf\n");

    for (i = 1; i < N - 1; i++)
    {
        for (j = 1; j < N - 1; j++)
        {
            fprintf(fileStaticPlot, "%lf\t%lf\t%lf\n", i * dx, j * dy, wave[i * N + j]);
        }
    }

    fclose(fileStaticPlot);
}

void initialCondition(double *wave)
{
    int i, j;
    for (j = 0; j < N; j++)
    {
        for (i = 0; i < N; i++)
        {
            wave[i * N + j] = 4 * sin(M_PI * i / 75.0);
        }
    }
}

void derivativeCondition(double *wave, double *wavePast){

	int i, j;

	for (i = 1; i < N-1; i++)
	{
		for (j = 1; j < N-1; j++)
		{
			wave[i*N+j] = (2*wavePast[i*N + j]*(1 - alpha*alpha - gamma*gamma) + alpha*alpha*wavePast[(i+1)*N + j] + alpha*alpha*wavePast[(i-1)*N + j] + gamma*gamma*wavePast[i*N + (j+1)] +  gamma*gamma*wavePast[i*N + (j-1)])/2;
		}
	}
}

void deviceCapabilities()
{

    hipDeviceProp_t prop;
    int count;
    hipGetDeviceCount(&count);
    for (int i = 0; i < count; i++)
    {
        hipGetDeviceProperties(&prop, i);
        printf(" --- General Information for device %d ---\n", i);
        printf("Name: %s\n", prop.name);
        printf("Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("Clock rate: %d\n", prop.clockRate);
        printf("Device copy overlap: ");
        if (prop.deviceOverlap)
            printf("Enabled\n");
        else
            printf("Disabled\n");
        printf("Kernel execition timeout : ");
        if (prop.kernelExecTimeoutEnabled)
            printf("Enabled\n");
        else
            printf("Disabled\n");
        printf(" --- Memory Information for device %d ---\n", i);
        printf("Total global mem: %ld\n", prop.totalGlobalMem);
        printf("Total constant Mem: %ld\n", prop.totalConstMem);
        printf("Max mem pitch: %ld\n", prop.memPitch);
        printf("Texture Alignment: %ld\n", prop.textureAlignment);
        printf(" --- MP Information for device %d ---\n", i);
        printf("Multiprocessor count: %d\n",
               prop.multiProcessorCount);
        printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
        printf("Registers per mp: %d\n", prop.regsPerBlock);
        printf("Threads in warp: %d\n", prop.warpSize);
        printf("Max threads per block: %d\n",
               prop.maxThreadsPerBlock);
        printf("Max thread dimensions: (%d, %d, %d)\n",
               prop.maxThreadsDim[0], prop.maxThreadsDim[1],
               prop.maxThreadsDim[2]);
        printf("Max grid dimensions: (%d, %d, %d)\n",
               prop.maxGridSize[0], prop.maxGridSize[1],
               prop.maxGridSize[2]);
        printf("\n");
    }
}

void actionWork(double dx, double dy){

    int i, j, k;

    double *hostWave, *hostWaveFuture, *hostWavePast;       // Host variables
    double *deviceWave, *deviceWaveFuture, *deviceWavePast; // Device Variables

    printf("Alocando memoria no host\n");
    hostWave = (double *)calloc((N * N), sizeof(double));
    hostWaveFuture = (double *)calloc((N * N), sizeof(double));
    hostWavePast = (double *)calloc((N * N), sizeof(double));

    printf("Colocando condição inicial.\n");
    initialCondition(hostWavePast);
    printf("Colocando condição da derivada.\n"); 
    derivativeCondition(hostWave, hostWavePast);

    printf("Alocando memoria no Device\n");
    CHECK(hipMalloc(&deviceWave, (N * N) * sizeof(double)));
    CHECK(hipMalloc(&deviceWaveFuture, (N * N) * sizeof(double)));
    CHECK(hipMalloc(&deviceWavePast, (N * N) * sizeof(double))); 
    
    printf("Iniciando calculo da função de onda.\n");
    for (i = 0; i < tempoTotal; i++)
    {
        CHECK(hipMemcpy(deviceWave, hostWave, (N * N) * sizeof(double), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(deviceWaveFuture, hostWaveFuture, (N * N) * sizeof(double), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(deviceWavePast, hostWavePast, (N * N) * sizeof(double), hipMemcpyHostToDevice));

        waveEquationKernel<<<N, N>>>(deviceWave, deviceWaveFuture, deviceWavePast);

        CHECK(hipMemcpy(hostWave, deviceWave, (N * N) * sizeof(double), hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(hostWaveFuture, deviceWaveFuture, (N * N) * sizeof(double), hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(hostWavePast, deviceWavePast, (N * N) * sizeof(double), hipMemcpyDeviceToHost));

        for (k = 1; k < N - 1; k++)
        {
            for (j = 1; j < N - 1; j++)
            {
                hostWavePast[k * N + j] = hostWave[k * N + j];
                hostWave[k * N + j] = hostWaveFuture[k * N + j];
            }
        }
    }

    printf("Escrevendo no arquivo o resultado do cálculo\n");
    writeFiles(hostWave, dx, dy);

    printf("Liberando memoria no host e device ... \n");

    free(hostWave);
    free(hostWaveFuture);
    free(hostWavePast);

    hipFree(deviceWave);
    hipFree(deviceWaveFuture);
    hipFree(deviceWavePast);
}

int main()
{
    double dx, dy;

    dx = (xFinal - xInicial) / N;
    dy = (yFinal - yInicial) / N;

    deviceCapabilities();

    clock_t beginTime = clock();

    actionWork(dx, dy);

    clock_t endTime = clock();

    printf("Time: %10.2f seconds \n", (endTime - beginTime) / (1.0 * CLOCKS_PER_SEC));
    return 0;
}
