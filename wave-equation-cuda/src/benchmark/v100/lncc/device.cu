
#include "hip/hip_runtime.h"
#include ""
#include <malloc.h>
#include <stdio.h>

#define CHECK(call)                                                \
    {                                                              \
        hipError_t error = call;                                  \
        if (error != hipSuccess)                                  \
        {                                                          \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
        }                                                          \
    }

void deviceCapabilities()
{

    hipDeviceProp_t prop;
    int count;
    hipGetDeviceCount(&count);
    for (int i = 0; i < count; i++)
    {
        hipGetDeviceProperties(&prop, i);
        printf("\n --- General Information for device %d ---\n", i);
        printf("Name: %s\n", prop.name);
        printf("Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("Clock rate: %d\n", prop.clockRate);
        printf("Device copy overlap: ");
        if (prop.deviceOverlap)
            printf("Enabled\n");
        else
            printf("Disabled\n");
        printf("Kernel execition timeout : ");
        if (prop.kernelExecTimeoutEnabled)
            printf("Enabled\n");
        else
            printf("Disabled\n");
        printf("\n --- Memory Information for device %d ---\n", i);
        printf("Total global mem: %ld\n", prop.totalGlobalMem);
        printf("Total constant Mem: %ld\n", prop.totalConstMem);
        printf("Max mem pitch: %ld\n", prop.memPitch);
        printf("Texture Alignment: %ld\n", prop.textureAlignment);
        printf("\n --- MP Information for device %d ---\n", i);
        printf("Multiprocessor count: %d\n",
               prop.multiProcessorCount);
        printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
        printf("Registers per mp: %d\n", prop.regsPerBlock);
        printf("Threads in warp: %d\n", prop.warpSize);
        printf("Max threads per block: %d\n",
               prop.maxThreadsPerBlock);
        printf("Max thread dimensions: (%d, %d, %d)\n",
               prop.maxThreadsDim[0], prop.maxThreadsDim[1],
               prop.maxThreadsDim[2]);
        printf("Max grid dimensions: (%d, %d, %d)\n",
               prop.maxGridSize[0], prop.maxGridSize[1],
               prop.maxGridSize[2]);
        printf("\n\n");
    }
}

int main()
{
    deviceCapabilities();
}